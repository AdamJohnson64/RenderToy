#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// RenderToy - A bit of history that's now a bit of silicon...
// Copyright (C) Adam Johnson 2017
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// This file contains the core CUDA implementation of the raytracer.
////////////////////////////////////////////////////////////////////////////////

#pragma region - Section : Boilerplate & Platform -
#include <hip/hip_runtime.h>
#include <math.h>
#include <cfloat>
#include <functional>
#include "Raytrace.h"

namespace RaytraceCUDA {
	#define DEVICE_PREFIX __device__
	#define DEVICE_SUFFIX
	#include "Raytrace.inc"
	#undef DEVICE_SUFFIX
	#undef DEVICE_PREFIX
}

void CUDA_CALL(hipError_t error) {
	if (error == 0) return;
}

#define TRY_CUDA(fn) CUDA_CALL(fn);
#pragma endregion

#pragma region - Render Mode : Common -
template <typename FLOAT, typename T, int X_SUPERSAMPLES = 1, int Y_SUPERSAMPLES = 1>
__device__ void cudaFill(const Scene<FLOAT>& pScene, Matrix44<FLOAT> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	RaytraceCUDA::PixelSetARGB<FLOAT> setpixel(bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, x, y);
	RaytraceCUDA::ComputePixel<FLOAT, T>(pScene, inverse_mvp, setpixel, X_SUPERSAMPLES, Y_SUPERSAMPLES);
}

template <typename FLOAT>
void cudaRender(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, std::function<void(const Scene<FLOAT>&, const Matrix44<FLOAT>&, void*, int, int, const dim3&, const dim3&)> fn) {
	// Allocate the scene buffer for CUDA.
	Scene<FLOAT>* host_scene = (Scene<FLOAT>*)pScene;
	Scene<FLOAT>* device_scene = nullptr;
	TRY_CUDA(hipMalloc((void**)&device_scene, host_scene->FileSize));
	TRY_CUDA(hipMemcpy(device_scene, host_scene, host_scene->FileSize, hipMemcpyHostToDevice));
	// Allocate the bitmap buffer for CUDA.
	void* device_bitmap_ptr = nullptr;
	int device_bitmap_stride = 4 * bitmap_width;
	TRY_CUDA(hipMalloc((void **)&device_bitmap_ptr, device_bitmap_stride * bitmap_height));
	// Launch the kernel.
	dim3 grid((bitmap_width + 15) / 16, (bitmap_height + 15) / 16, 1);
	dim3 threads(16, 16, 1);
	fn(*device_scene, *(Matrix44<FLOAT>*)pInverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, grid, threads);
	// Copy back the render result to the CPU buffer.
	for (int y = 0; y < bitmap_height; ++y)
	{
		void* pDevice = (unsigned char*)device_bitmap_ptr + device_bitmap_stride * y;
		void* pHost = (unsigned char*)bitmap_ptr + bitmap_stride * y;
		TRY_CUDA(hipMemcpy(pHost, pDevice, 4 * bitmap_width, hipMemcpyDeviceToHost));
	}
	// Clean up.
	TRY_CUDA(hipFree(device_bitmap_ptr));
	device_bitmap_ptr = nullptr;
	TRY_CUDA(hipFree(device_scene));
	device_scene = nullptr;
}
#pragma endregion

#pragma region - Render Mode : Raycast -
__global__ void cudaRaycastKernelF32(const Scene<float>& pScene, Matrix44<float> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<float, RaytraceCUDA::RenderModeRaycast<float>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastCUDAF32(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<float>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<float>& device_scene, const Matrix44<float>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastKernelF32<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}

__global__ void cudaRaycastKernelF64(const Scene<double>& pScene, Matrix44<double> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<double, RaytraceCUDA::RenderModeRaycast<double>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastCUDAF64(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<double>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<double>& device_scene, const Matrix44<double>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastKernelF64<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}
#pragma endregion

#pragma region - Render Mode : Raycast Bitangents -
__global__ void cudaRaycastBitangentsKernelF32(const Scene<float>& pScene, Matrix44<float> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<float, RaytraceCUDA::RenderModeRaycastBitangents<float>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastBitangentsCUDAF32(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<float>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<float>& device_scene, const Matrix44<float>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastBitangentsKernelF32<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}

__global__ void cudaRaycastBitangentsKernelF64(const Scene<double>& pScene, Matrix44<double> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<double, RaytraceCUDA::RenderModeRaycastBitangents<double>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastBitangentsCUDAF64(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<double>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<double>& device_scene, const Matrix44<double>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastBitangentsKernelF64<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}
#pragma endregion

#pragma region - Render Mode : Raycast Normals -
__global__ void cudaRaycastNormalsKernelF32(const Scene<float>& pScene, Matrix44<float> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<float, RaytraceCUDA::RenderModeRaycastNormals<float>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastNormalsCUDAF32(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<float>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<float>& device_scene, const Matrix44<float>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastNormalsKernelF32<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}

__global__ void cudaRaycastNormalsKernelF64(const Scene<double>& pScene, Matrix44<double> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<double, RaytraceCUDA::RenderModeRaycastNormals<double>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastNormalsCUDAF64(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<double>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<double>& device_scene, const Matrix44<double>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastNormalsKernelF64<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}
#pragma endregion

#pragma region - Render Mode : Raycast Tangents -
__global__ void cudaRaycastTangentsKernelF32(const Scene<float>& pScene, Matrix44<float> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<float, RaytraceCUDA::RenderModeRaycastTangents<float>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastTangentsCUDAF32(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<float>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<float>& device_scene, const Matrix44<float>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastTangentsKernelF32<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}

__global__ void cudaRaycastTangentsKernelF64(const Scene<double>& pScene, Matrix44<double> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<double, RaytraceCUDA::RenderModeRaycastTangents<double>>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaycastTangentsCUDAF64(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<double>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<double>& device_scene, const Matrix44<double>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaycastTangentsKernelF64<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}
#pragma endregion

#pragma region - Render Mode : Raytrace -
__global__ void cudaRaytraceKernelF32(const Scene<float>& pScene, Matrix44<float> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<float, RaytraceCUDA::RenderModeRaytrace<float>, 2, 2>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaytraceCUDAF32(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<float>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<float>& device_scene, const Matrix44<float>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaytraceKernelF32<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}

__global__ void cudaRaytraceKernelF64(const Scene<double>& pScene, Matrix44<double> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride) {
	cudaFill<double, RaytraceCUDA::RenderModeRaytrace<double>, 2, 2>(pScene, inverse_mvp, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride);
}

extern "C" void RaytraceCUDAF64(const void* pScene, const void* pInverseMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	cudaRender<double>(pScene, pInverseMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, [](const Scene<double>& device_scene, const Matrix44<double>& InverseMVP, void* device_bitmap_ptr, int bitmap_width, int bitmap_height, const dim3& grid, const dim3& threads) {
		cudaRaytraceKernelF64<<<grid, threads>>>(device_scene, InverseMVP, device_bitmap_ptr, bitmap_width, bitmap_height, 4 * bitmap_width);
	});
}
#pragma endregion

#pragma region - Render Mode : Ambient Occlusion -
template <typename FLOAT>
__global__ void cudaAOC(const Scene<FLOAT>& pScene, Matrix44<FLOAT> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const Vector4<FLOAT>* hemisamples) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	RaytraceCUDA::PixelSetARGB<FLOAT> setpixel(bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, x, y);
	RaytraceCUDA::ComputePixelAOC<FLOAT>(pScene, inverse_mvp, setpixel, hemisample_count, hemisamples);
}

template <typename FLOAT>
void AmbientOcclusionCUDA(const void* pScene, const void* pMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const void* hemisamples)
{
	// Allocate the scene buffer for CUDA.
	Scene<FLOAT>* host_scene = (Scene<FLOAT>*)pScene;
	Scene<FLOAT>* device_scene = nullptr;
	TRY_CUDA(hipMalloc((void**)&device_scene, host_scene->FileSize));
	TRY_CUDA(hipMemcpy(device_scene, host_scene, host_scene->FileSize, hipMemcpyHostToDevice));
	// Allocate the bitmap buffer for CUDA.
	void* device_bitmap_ptr = nullptr;
	int device_bitmap_stride = 4 * bitmap_width;
	TRY_CUDA(hipMalloc((void**)&device_bitmap_ptr, device_bitmap_stride * bitmap_height));
	// Allocate the hemisample buffer for CUDA.
	Vector4<FLOAT>* device_hemisamples = nullptr;
	TRY_CUDA(hipMalloc((void**)&device_hemisamples, sizeof(Vector4<FLOAT>) * hemisample_count));
	TRY_CUDA(hipMemcpy(device_hemisamples, hemisamples, sizeof(Vector4<FLOAT>) * hemisample_count, hipMemcpyHostToDevice));
	// Launch the kernel.
	dim3 grid((bitmap_width + 15) / 16, (bitmap_height + 15) / 16, 1);
	dim3 threads(16, 16, 1);
	cudaAOC<<<grid, threads>>>(*device_scene, *(Matrix44<FLOAT>*)pMVP, device_bitmap_ptr, bitmap_width, bitmap_height, device_bitmap_stride, hemisample_count, device_hemisamples);
	// Copy back the render result to the CPU buffer.
	for (int y = 0; y < bitmap_height; ++y)
	{
		void* pDevice = (unsigned char*)device_bitmap_ptr + device_bitmap_stride * y;
		void* pHost = (unsigned char*)bitmap_ptr + bitmap_stride * y;
		TRY_CUDA(hipMemcpy(pHost, pDevice, 4 * bitmap_width, hipMemcpyDeviceToHost));
	}
	// Clean up.
	TRY_CUDA(hipFree(device_hemisamples));
	device_hemisamples = nullptr;
	TRY_CUDA(hipFree(device_bitmap_ptr));
	device_bitmap_ptr = nullptr;
	TRY_CUDA(hipFree(device_scene));
	device_scene = nullptr;
}

extern "C" void AmbientOcclusionCUDAF32(const void* pScene, const void* pMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const void* hemisamples)
{
	AmbientOcclusionCUDA<float>(pScene, pMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, hemisample_count, hemisamples);
}

extern "C" void AmbientOcclusionCUDAF64(const void* pScene, const void* pMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const void* hemisamples)
{
	AmbientOcclusionCUDA<double>(pScene, pMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, hemisample_count, hemisamples);
}
#pragma endregion

#pragma region - Render Mode : Ambient Occlusion (Multipass) -
template <typename FLOAT>
__global__ void globalRescaleVec4(const Vector4<FLOAT>* acc_ptr, int acc_stride, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, FLOAT scale) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= bitmap_width || y >= bitmap_height) return;
	Vector4<FLOAT>* pRaster_Accum = (Vector4<FLOAT>*)((unsigned char*)acc_ptr + acc_stride * y);
	unsigned int* pRaster_Bitmap = (unsigned int*)((unsigned char*)bitmap_ptr + bitmap_stride * y);
	Vector4<FLOAT>* pPixel_Accum = pRaster_Accum + x;
	unsigned int* pPixel_Bitmap = pRaster_Bitmap + x;
	Vector4<FLOAT> result = RaytraceCUDA::make_vector4(pPixel_Accum->x * scale, pPixel_Accum->y * scale, pPixel_Accum->z * scale, pPixel_Accum->w * scale);
	*pPixel_Bitmap = RaytraceCUDA::Vector4ToA8R8G8B8(result);
}

template <typename FLOAT>
__global__ void globalAmbientOcclusionMPCUDA(const Scene<FLOAT>& pScene, Matrix44<FLOAT> inverse_mvp, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const Vector4<FLOAT>* hemisamples) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	RaytraceCUDA::PixelAccumulateVec4<FLOAT> setpixel(bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, x, y);
	RaytraceCUDA::ComputePixelAOC<FLOAT>(pScene, inverse_mvp, setpixel, hemisample_count, hemisamples);
}

template <typename FLOAT>
void AmbientOcclusionMPCUDA(const void* pScene, const void* pMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const void* hemisamples)
{
	// Allocate the scene buffer for CUDA.
	Scene<FLOAT>* host_scene = (Scene<FLOAT>*)pScene;
	Scene<FLOAT>* device_scene = nullptr;
	TRY_CUDA(hipMalloc((void**)&device_scene, host_scene->FileSize));
	TRY_CUDA(hipMemcpy(device_scene, host_scene, host_scene->FileSize, hipMemcpyHostToDevice));
	// Allocate the accumulation buffer for CUDA.
	Vector4<FLOAT>* device_acc_ptr = nullptr;
	int device_acc_stride = sizeof(Vector4<FLOAT>) * bitmap_width;
	TRY_CUDA(hipMalloc((void**)&device_acc_ptr, device_acc_stride * bitmap_height));
	// Allocate the hemisample buffer for CUDA.
	Vector4<FLOAT>* device_hemisamples = nullptr;
	TRY_CUDA(hipMalloc((void**)&device_hemisamples, sizeof(Vector4<FLOAT>) * hemisample_count));
	TRY_CUDA(hipMemcpy(device_hemisamples, hemisamples, sizeof(Vector4<FLOAT>) * hemisample_count, hipMemcpyHostToDevice));
	// Launch the accumulator kernel.
	int pass_hemisample_count = 64;
	int pass_count = hemisample_count / pass_hemisample_count;
	{
		dim3 grid((bitmap_width + 15) / 16, (bitmap_height + 15) / 16, 1);
		dim3 threads(16, 16, 1);
		for (int pass = 0; pass < pass_count; ++pass) {
			Vector4<FLOAT>* pass_device_hemisamples = device_hemisamples + pass * pass_hemisample_count;
			globalAmbientOcclusionMPCUDA << <grid, threads >> >(*device_scene, *(Matrix44<FLOAT>*)pMVP, device_acc_ptr, bitmap_width, bitmap_height, device_acc_stride, pass_hemisample_count, pass_device_hemisamples);
		}
	}
	// Allocate the tonemap buffer for CUDA.
	void* device_bitmap_ptr = nullptr;
	int device_bitmap_stride = sizeof(int) * bitmap_width;
	TRY_CUDA(hipMalloc((void**)&device_bitmap_ptr, device_bitmap_stride * bitmap_height));
	// Apply the tonemap and divide the accumulated buffer.
	{
		dim3 grid((bitmap_width + 15) / 16, (bitmap_height + 15) / 16, 1);
		dim3 threads(16, 16, 1);
		globalRescaleVec4<<<grid, threads>>>(device_acc_ptr, device_acc_stride, device_bitmap_ptr, bitmap_width, bitmap_height, device_bitmap_stride, FLOAT(1) / pass_count);
	}
	// Copy back the render result to the CPU buffer.
	for (int y = 0; y < bitmap_height; ++y)
	{
		void* pDevice = (unsigned char*)device_bitmap_ptr + device_bitmap_stride * y;
		void* pHost = (unsigned char*)bitmap_ptr + bitmap_stride * y;
		TRY_CUDA(hipMemcpy(pHost, pDevice, 4 * bitmap_width, hipMemcpyDeviceToHost));
	}
	// Clean up.
	TRY_CUDA(hipFree(device_bitmap_ptr));
	device_bitmap_ptr = nullptr;
	TRY_CUDA(hipFree(device_hemisamples));
	device_hemisamples = nullptr;
	TRY_CUDA(hipFree(device_acc_ptr));
	device_bitmap_ptr = nullptr;
	TRY_CUDA(hipFree(device_scene));
	device_scene = nullptr;
}

extern "C" void AmbientOcclusionMPCUDAF32(const void* pScene, const void* pMVP, void* bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride, int hemisample_count, const void* hemisamples)
{
	AmbientOcclusionMPCUDA<float>(pScene, pMVP, bitmap_ptr, bitmap_width, bitmap_height, bitmap_stride, hemisample_count, hemisamples);
}
#pragma endregion