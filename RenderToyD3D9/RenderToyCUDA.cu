#include <hip/hip_runtime.h>
#include <math.h>

////////////////////////////////////////////////////////////////////////////////
// Device Code.
////////////////////////////////////////////////////////////////////////////////

// Basic Math Primitives.
__device__ double3 operator-(const double3& val) { return make_double3(-val.x, -val.y, -val.z); }
__device__ double4 operator-(const double4& val) { return make_double4(-val.z, -val.y, -val.z, -val.w); }
__device__ double3 operator+(const double3& lhs, const double3& rhs) { return make_double3(lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z); }
__device__ double4 operator+(const double4& lhs, const double4& rhs) { return make_double4(lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z, lhs.w + rhs.w); }
__device__ double3 operator-(const double3& lhs, const double3& rhs) { return make_double3(lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z); }
__device__ double4 operator-(const double4& lhs, const double4& rhs) { return make_double4(lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z, lhs.w - rhs.w); }
__device__ double3 operator*(const double3& lhs, double rhs) { return make_double3(lhs.x * rhs, lhs.y * rhs, lhs.z * rhs); }
__device__ double3 operator*(double lhs, const double3& rhs) { return rhs * lhs; }
__device__ double4 operator*(const double4& lhs, double rhs) { return make_double4(lhs.x * rhs, lhs.y * rhs, lhs.z * rhs, lhs.w * rhs); }
__device__ double4 operator*(double lhs, const double4& rhs) { return rhs * lhs; }
__device__ double3 operator/(const double3 &lhs, double rhs) { return lhs * (1 / rhs); }
__device__ double4 operator/(const double4 &lhs, double rhs) { return lhs * (1 / rhs); }

// Common Math Primitives.
__device__ double Clamp(double min, double max, double val) { return val < min ? min : (val > max ? max : val); }
__device__ double Dot(const double3& lhs, const double3& rhs) { return lhs.x * rhs.x + lhs.y * rhs.y + lhs.z * rhs.z; }
__device__ double Length(const double3& val) { return norm3d(val.x, val.y, val.z); }
__device__ double Lerp(double y1, double y2, double x) { return y1 + (y2 - y1) * x; }
__device__ double3 Normalize(const double3 &val) { return val * rnorm3d(val.x, val.y, val.z); }

// Matrix Math.
struct Matrix4D { double M[16]; };

__device__ Matrix4D CreateMatrixIdentity() {
	double m[] = { 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1 };
	return *(Matrix4D*)m;
}

__device__ Matrix4D CreateMatrixTranslate(double x, double y, double z) {
	double m[] = { 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, x, y, z, 1 };
	return *(Matrix4D*)m;
}

__device__ double3 TransformPoint(const Matrix4D& m, const double3& p) {
	return make_double3(
		m.M[0] * p.x + m.M[4] * p.y + m.M[8] * p.z + m.M[12],
		m.M[1] * p.x + m.M[5] * p.y + m.M[9] * p.z + m.M[13],
		m.M[2] * p.x + m.M[6] * p.y + m.M[10] * p.z + m.M[14]);
}

__device__ double3 TransformVector(const Matrix4D& m, const double3& p) {
	return make_double3(
		m.M[0] * p.x + m.M[4] * p.y + m.M[8] * p.z,
		m.M[1] * p.x + m.M[5] * p.y + m.M[9] * p.z,
		m.M[2] * p.x + m.M[6] * p.y + m.M[10] * p.z);
}

__device__ double4 Transform(const Matrix4D& m, const double4& p) {
	return make_double4(
		m.M[0] * p.x + m.M[4] * p.y + m.M[8] * p.z + m.M[12] * p.w,
		m.M[1] * p.x + m.M[5] * p.y + m.M[9] * p.z + m.M[13] * p.w,
		m.M[2] * p.x + m.M[6] * p.y + m.M[10] * p.z + m.M[14] * p.w,
		m.M[3] * p.x + m.M[7] * p.y + m.M[11] * p.z + m.M[15] * p.w);
}

__device__ double3 Reflect(const double3& incident, const double3& normal) {
	return incident - 2 * Dot(incident, normal) * normal;
}

__device__ double3 Refract(const double3& incident, const double3& normal, double ior) {
	double cosi = Clamp(-1, 1, Dot(incident, normal));
	double etai = 1, etat = ior;
	double3 n = normal;
	if (cosi < 0) { cosi = -cosi; }
	else { double tmp = etai; etai = etat; etat = tmp; n = -normal; }
	double eta = etai / etat;
	double k = 1 - eta * eta * (1 - cosi * cosi);
	return k < 0 ? make_double3(0, 0, 0) : (eta * incident + (eta * cosi - sqrt(k)) * n);
}

__device__ double SchlickApprox(const double3& incident, const double3 &normal, double ior1, double ior2) {
	double coeff = (ior1 - ior2) / (ior1 + ior2);
	coeff = coeff * coeff;
	return coeff + (1 - coeff) * pow((1 - Dot(-incident, normal)), 5);
}

// Geometric Math.
__device__ double IntersectPlane(const double3 &origin, const double3 &direction) {
	const double PLANE_DISTANCE = 0;
	const double3 PLANE_NORMAL = make_double3(0, 1, 0);
	return (PLANE_DISTANCE - Dot(PLANE_NORMAL, origin)) / Dot(PLANE_NORMAL, direction);
}

__device__ double IntersectSphere(const double3 &origin, const double3 &direction) {
	const double SPHERE_RADIUS = 1;
	double a = Dot(direction, direction);
	double b = 2 * Dot(origin, direction);
	double c = Dot(origin, origin) - SPHERE_RADIUS * SPHERE_RADIUS;
	// If the determinant is negative then there are no real roots and this will be NaN.
	double det = sqrt(b * b - 4 * a * c);
	// "a" cannot be negative so (worst case) these lambdas are +Inf.
	double den = 2 * a;
	double lambda1 = (-b - det) / den;
	double lambda2 = (-b + det) / den;
	double lambda_best = 1000000;
	if (lambda1 >= 0 && lambda1 < lambda_best) lambda_best = lambda1;
	if (lambda2 >= 0 && lambda2 < lambda_best) lambda_best = lambda2;
	return lambda_best;
}

enum GeometryType {
	GEOMETRY_NONE = 0,
	GEOMETRY_PLANE = 1,
	GEOMETRY_SPHERE = 2
};

enum MaterialType {
	MATERIAL_NONE = 0,
	MATERIAL_CHECKERBOARD_XZ = 1,
	MATERIAL_RED,
	MATERIAL_GREEN,
	MATERIAL_BLUE,
	MATERIAL_GLASS
};

__device__ double Intersect(const double3 &origin, const double3 &direction, GeometryType geometry) {
	switch (geometry) {
	case GEOMETRY_PLANE:
		return IntersectPlane(origin, direction);
	case GEOMETRY_SPHERE:
		return IntersectSphere(origin, direction);
	default: return 1000000;
	}
}

struct SceneObject {
	Matrix4D Transform;
	Matrix4D TransformInverse;
	GeometryType Geometry;
	MaterialType Material;
};

const int SCENE_COUNT = 4;

// Raytracer Ray Tests.
__device__ bool RayShadow(const double3 &origin, const double3 &direction) {
	const SceneObject TheScene[] = {
		SceneObject{ CreateMatrixIdentity(), CreateMatrixIdentity(), GEOMETRY_PLANE, MATERIAL_CHECKERBOARD_XZ },
		SceneObject{ CreateMatrixTranslate(-2, 1, 0), CreateMatrixTranslate(2, -1, 0), GEOMETRY_SPHERE, MATERIAL_RED },
		SceneObject{ CreateMatrixTranslate(0, 1, 0), CreateMatrixTranslate(0, -1, 0), GEOMETRY_SPHERE, MATERIAL_GREEN },
		SceneObject{ CreateMatrixTranslate(2, 1, 0), CreateMatrixTranslate(-2, -1, 0), GEOMETRY_SPHERE, MATERIAL_BLUE },
	};
	for (int i = 0; i < SCENE_COUNT; ++i) {
		const SceneObject &scene_object = TheScene[i];
		double3 transformed_origin = TransformPoint(scene_object.TransformInverse, origin);
		double3 transformed_direction = TransformVector(scene_object.TransformInverse, direction);
		double lambda = Intersect(transformed_origin, transformed_direction, scene_object.Geometry);
		if (lambda >= 0 && lambda < 1000000) return true;
	}
	return false;
}

template <int RECURSE>
__device__ double4 RayColor(const double3 &origin, const double3 &direction) {
	// Start intersecting objects.
	const SceneObject TheScene[] = {
		SceneObject{ CreateMatrixIdentity(), CreateMatrixIdentity(), GEOMETRY_PLANE, MATERIAL_CHECKERBOARD_XZ },
		SceneObject{ CreateMatrixTranslate(-2, 1, 0), CreateMatrixTranslate(2, -1, 0), GEOMETRY_SPHERE, MATERIAL_RED },
		SceneObject{ CreateMatrixTranslate(0, 1, 0), CreateMatrixTranslate(0, -1, 0), GEOMETRY_SPHERE, MATERIAL_GREEN },
		SceneObject{ CreateMatrixTranslate(2, 1, 0), CreateMatrixTranslate(-2, -1, 0), GEOMETRY_SPHERE, MATERIAL_BLUE },
	};
	double best_lambda = 1000000;
	int best_pobject = 0;
	for (int i = 0; i < SCENE_COUNT; ++i) {
		const SceneObject &scene_object = TheScene[i];
		double3 transformed_origin = TransformPoint(scene_object.TransformInverse, origin);
		double3 transformed_direction = TransformVector(scene_object.TransformInverse, direction);
		double lambda = Intersect(transformed_origin, transformed_direction, scene_object.Geometry);
		if (lambda >= 0 && lambda < best_lambda) {
			best_lambda = lambda;
			best_pobject = i;
		}
	}
	if (best_lambda == 1000000) {
		return make_double4(0, 0, 0, 0);
	}
	{
		double3 p = origin + best_lambda * direction;
		double3 l = Normalize(make_double3(10, 10, -10) - p);
		double3 n;
		switch (TheScene[best_pobject].Geometry) {
		case GEOMETRY_PLANE:
			n = make_double3(0, 1, 0);
			break;
		case GEOMETRY_SPHERE:
			n = TransformPoint(TheScene[best_pobject].TransformInverse, p);
			break;
		default:
			n = make_double3(0, 1, 0);
		}
		double3 i = Normalize(direction);
		double3 r = Reflect(i, n);
		double3 t = Refract(i, n, 1.5);
		double scale_diffuse = Dot(l, n);
		scale_diffuse = scale_diffuse > 0 ? scale_diffuse : 0;
		double scale_specular = Dot(l, r);
		scale_specular = scale_specular > 0 ? scale_specular : 0;
		scale_specular = pow(scale_specular, 100);
		if (RayShadow(p + 0.0001 * n, l)) scale_diffuse *= 0.5;
		double4 color_diffuse;
		double4 color_specular;
		double scale_reflect;
		double scale_refract;
		switch (TheScene[best_pobject].Material) {
		case MATERIAL_CHECKERBOARD_XZ:
		{
			int mx = (p.x - floor(p.x)) < 0.5 ? 0 : 1;
			int my = 0; // (space.Y - floor(space.Y)) < 0.5 ? 0 : 1;
			int mz = (p.z - floor(p.z)) < 0.5 ? 0 : 1;
			double c = (mx + my + mz) % 2;
			color_diffuse = make_double4(c, c, c, 0);
			color_specular = make_double4(1, 1, 1, 0);
			scale_reflect = 1;
			scale_refract = 0;
		}
		break;
		case MATERIAL_RED:
		{
			//color_diffuse = make_double4(1, 0, 0, 0);
			color_diffuse = make_double4(0, 0, 0, 0);
			color_specular = make_double4(1, 1, 1, 0);
			scale_reflect = 0;
			scale_refract = 1;
		}
		break;
		case MATERIAL_GREEN:
		{
			color_diffuse = make_double4(0, 0.5, 0, 0);
			color_specular = make_double4(1, 1, 1, 0);
			scale_reflect = 0.5;
			scale_refract = 0;
		}
		break;
		case MATERIAL_BLUE:
		{
			color_diffuse = make_double4(0, 0, 1, 0);
			color_specular = make_double4(1, 1, 1, 0);
			scale_reflect = 0.5;
			scale_refract = 0;
		}
		break;
		case MATERIAL_GLASS:
		{
			color_diffuse = make_double4(0, 0, 0, 0);
			color_specular = make_double4(1, 1, 1, 0);
			scale_reflect = 0;
			scale_refract = 1;
		}
		break;
		}
		double4 color = color_diffuse * scale_diffuse + color_specular * scale_specular;
		if (scale_refract > 0) {
			//Schlick's Approximation for reflectance.
			double R = SchlickApprox(i, n, 1, 1.5);
			scale_reflect = R;
			//Add refraction contribution.
			color = color + RayColor<RECURSE - 1>(p + t * 0.0001, t) * scale_refract;
		}
		if (scale_reflect > 0) {
			color = color + RayColor<RECURSE - 1>(p + r * 0.0001, r) * scale_reflect;
		}
		color.w = 1;
		return color;
	}
}

template <>
__device__ double4 RayColor<-1>(const double3 &origin, const double3 &direction) {
	return make_double4(0, 0, 0, 0);
}

__device__ double4 RayColor(const double3 &origin, const double3 &direction) {
	return RayColor<2>(origin, direction);
}

__device__ void ComputeRay(const Matrix4D &inverse_mvp, double clipx, double clipy, double3 &origin, double3 &direction) {
	double4 v41 = Transform(inverse_mvp, make_double4(clipx, clipy, 0, 1));
	double4 v42 = Transform(inverse_mvp, make_double4(clipx, clipy, 1, 1));
	double3 ray_p1 = make_double3(v41.x / v41.w, v41.y / v41.w, v41.z / v41.w);
	double3 ray_p2 = make_double3(v42.x / v42.w, v42.y / v42.w, v42.z / v42.w);
	origin = ray_p1;
	direction = ray_p2 - ray_p1;
}

__device__ unsigned int MakePixel(const double4 &color) {
	unsigned char r = color.x < 0 ? 0 : (color.x > 1 ? 1 : color.x) * 255;
	unsigned char g = color.y < 0 ? 0 : (color.y > 1 ? 1 : color.y) * 255;
	unsigned char b = color.z < 0 ? 0 : (color.z > 1 ? 1 : color.z) * 255;
	unsigned char a = color.w < 0 ? 0 : (color.w > 1 ? 1 : color.w) * 255;
	return (a << 24) | (r << 16) | (g << 8) | (b << 0);
}

__global__ void cudaRaytraceKernel(Matrix4D inverse_mvp, void *bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	// Generate untransformed ray.
	double3 origin;
	double3 direction;
	double4 color = make_double4(0, 0, 0, 0);
	const int X_SUPERSAMPLES = 1;
	const int Y_SUPERSAMPLES = 1;
	for (int y_supersample = 1; y_supersample <= Y_SUPERSAMPLES; ++y_supersample) {
		for (int x_supersample = 1; x_supersample <= X_SUPERSAMPLES; ++x_supersample) {
			// Build a ray for this supersample.
			double vx = Lerp(-1, +1, (x + x_supersample / (X_SUPERSAMPLES + 1.0)) / bitmap_width);
			double vy = Lerp(+1, -1, (y + y_supersample / (Y_SUPERSAMPLES + 1.0)) / bitmap_height);
			ComputeRay(inverse_mvp, vx, vy, origin, direction);
			// Compute intersection with plane.
			color = color + RayColor(origin, direction);
		}
	}
	color = color / (X_SUPERSAMPLES * Y_SUPERSAMPLES);
	// Fill in the pixel.
	void *pRaster = (unsigned char*)bitmap_ptr + bitmap_stride * y;
	void *pPixel = (unsigned char*)pRaster + 4 * x;
	*(unsigned int*)pPixel = MakePixel(color);
}

////////////////////////////////////////////////////////////////////////////////
// Host Code
////////////////////////////////////////////////////////////////////////////////

void CUDA_CALL(hipError_t error) {
	if (error == 0) return;
	int test = 0;
}

#define TRY_CUDA(fn) CUDA_CALL(fn);

extern "C" bool cudaRaytrace(double* pMVP, void *bitmap_ptr, int bitmap_width, int bitmap_height, int bitmap_stride)
{
	void *device_buffer = nullptr;
	int buffer_stride = 4 * bitmap_width;
	TRY_CUDA(hipMalloc((void **)&device_buffer, buffer_stride * bitmap_height));
	Matrix4D MVP = *(Matrix4D*)pMVP;
	dim3 grid(bitmap_width / 16, bitmap_height / 16, 1);
	dim3 threads(16, 16, 1);
	cudaRaytraceKernel<<<grid, threads>>>(MVP, device_buffer, bitmap_width, bitmap_height, 4 * bitmap_width);
	for (int y = 0; y < bitmap_height; ++y)
	{
		void* pDevice = (unsigned char*)device_buffer + buffer_stride * y;
		void* pHost = (unsigned char*)bitmap_ptr + bitmap_stride * y;
		TRY_CUDA(hipMemcpy(pHost, pDevice, 4 * bitmap_width, hipMemcpyDeviceToHost));
		int test = 0;
	}
	TRY_CUDA(hipFree(device_buffer));
	device_buffer = nullptr;
	return true;
}
